#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include ""

#include <stdio.h>
#include <stdlib.h>


extern "C"{
	void cu_RGBtoYUV444(unsigned char *yuv, unsigned char *pData, int width, int height);
	void cu_RGBtoYUV420Pitch(unsigned char *bufY, unsigned char *bufU, unsigned char *bufV, unsigned char *pData, int stride, int width, int height);
	void cu_UVtoNV12(unsigned char *pOut, unsigned char *bufU, unsigned char *bufV, int stride, int width, int height);
	void cu_customCudaMemcpy(unsigned char *yuv, unsigned char *pData, int width, int height, int bytes);
	void cu_RGBtoNV12(unsigned char *pDest, unsigned char *pSrc, int stride, int width, int height);
}
__global__ void convertRGBtoYUV444p(unsigned char *input, unsigned char *output, int dst_width, int dst_height);
__global__ void convertRGBtoYUV420pPitch(unsigned char *input, unsigned char *bufY, unsigned char *bufU, unsigned char *bufV, int stride, int dst_width, int dst_height);
__global__ void convertUVtoNV12(unsigned char *pOut, unsigned char *bufU, unsigned char *bufV, int stride, int width, int height);
__global__ void customCudaMemcpy(unsigned char *input, unsigned char *output, int dst_width, int dst_height, int bytes);
__global__ void convertRGBtoNV12(unsigned char *output, unsigned char *input, int stride, int dst_width, int dst_height);

void cu_RGBtoYUV444(unsigned char *yuv, unsigned char *pData, int width, int height)
{
	dim3 dimblock(8, 8);
	dim3 dimgrid(
		(width + dimblock.x - 1) / dimblock.x,
		(height + dimblock.y - 1) / dimblock.y);

	convertRGBtoYUV444p << <dimgrid, dimblock >> >(pData, yuv, width, height);
}

__global__ void convertRGBtoYUV444p(unsigned char *input, unsigned char *output, int dst_width, int dst_height)
{
	int RGBcomponentCount = 3;
	unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
	int dst_size = dst_width * dst_height;

	unsigned int R = input[(x + y * dst_width) * RGBcomponentCount];
	unsigned int G = input[(x + y * dst_width) * RGBcomponentCount + 1];
	unsigned int B = input[(x + y * dst_width) * RGBcomponentCount + 2];

	float Y = ((0.299 * R + 0.587 * G + 0.114 * B) * 219 / 255) + 16; if (Y >= 255) Y = 255;
	float U = ((-0.299 * R - 0.587 * G + 0.886 * B) * 224 / 1.772 / 255) + 128; if (U >= 255) U = 255;
	float V = ((0.701 * R - 0.587 * G - 0.114 * B) * 224 / 1.402 / 255) + 128; if (V >= 255) V = 255;

	output[x + y * dst_width] = (unsigned char)Y;
	output[dst_size + (x + y * dst_width)] = (unsigned char)U;
	output[(dst_size * 2) + (x + y * dst_width)] = (unsigned char)V;
}

void cu_customCudaMemcpy(unsigned char *yuv, unsigned char *pData, int width, int height, int bytes)
{
	dim3 dimblock(8, 8);
	dim3 dimgrid(
		(width + dimblock.x - 1) / dimblock.x,
		(height + dimblock.y - 1) / dimblock.y);

	customCudaMemcpy << <dimgrid, dimblock >> >(pData, yuv, width, height, bytes);
}

__global__ void customCudaMemcpy(unsigned char *input, unsigned char *output, int dst_width, int dst_height, int bytes)
{
	unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

	output[(x + y * dst_width) * bytes] = input[(x + y * dst_width) * bytes];
	output[(x + y * dst_width) * bytes + 1] = input[(x + y * dst_width) * bytes + 1];
	output[(x + y * dst_width) * bytes + 2] = input[(x + y * dst_width) * bytes + 2];
}

void cu_RGBtoNV12(unsigned char *pDest, unsigned char *pSrc, int stride, int width, int height)
{
	dim3 dimblock(8, 8);
	dim3 dimgrid(
		(width + dimblock.x - 1) / dimblock.x,
		(height + dimblock.y - 1) / dimblock.y);

	convertRGBtoNV12 << <dimgrid, dimblock >> >(pDest, pSrc, stride, width, height);
}

__global__ void convertRGBtoNV12(unsigned char *output, unsigned char *input, int stride, int dst_width, int dst_height)
{
	int RGBcomponentCount = 3;
	unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
	int dst_size = stride * dst_height;

	// load RGB values
	unsigned int R = input[(x + y * dst_width) * RGBcomponentCount];
	unsigned int G = input[(x + y * dst_width) * RGBcomponentCount + 1];
	unsigned int B = input[(x + y * dst_width) * RGBcomponentCount + 2];

	float Y = ((0.299 * R + 0.587 * G + 0.114 * B) * 219 / 255) + 16; if (Y >= 255) Y = 255;
	float U = ((-0.299 * R - 0.587 * G + 0.886 * B) * 224 / 1.772 / 255) + 128; if (U >= 255) U = 255;
	float V = ((0.701 * R - 0.587 * G - 0.114 * B) * 224 / 1.402 / 255) + 128; if (V >= 255) V = 255;

	output[x + y * stride] = Y;
	output[((x / 2) * 2) + ((y / 2) * stride) + (stride * dst_height)] = U;
	output[((x / 2) * 2) + 1 + ((y / 2) * stride) + (stride * dst_height)] = V;
}

void cu_RGBtoYUV420Pitch(unsigned char *bufY, unsigned char *bufU, unsigned char *bufV, unsigned char *pData, int stride, int width, int height)
{
	dim3 dimblock(8, 8);
	dim3 dimgrid(
		(width + dimblock.x - 1) / dimblock.x,
		(height + dimblock.y - 1) / dimblock.y);

	convertRGBtoYUV420pPitch << <dimgrid, dimblock >> >(pData, bufY, bufU, bufV, stride, width, height);
}

__global__ void convertRGBtoYUV420pPitch(unsigned char *input, unsigned char *bufY, unsigned char *bufU, unsigned char *bufV, int stride, int dst_width, int dst_height)
{
	int RGBcomponentCount = 3;
	unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
	int dst_size = stride * dst_height;

	// load RGB values
	unsigned int R = input[(x + y * dst_width) * RGBcomponentCount];
	unsigned int G = input[(x + y * dst_width) * RGBcomponentCount + 1];
	unsigned int B = input[(x + y * dst_width) * RGBcomponentCount + 2];

	float Y = ((0.299 * R + 0.587 * G + 0.114 * B) * 219 / 255) + 16; if (Y >= 255) Y = 255;
	float U = ((-0.299 * R - 0.587 * G + 0.886 * B) * 224 / 1.772 / 255) + 128; if (U >= 255) U = 255;
	float V = ((0.701 * R - 0.587 * G - 0.114 * B) * 224 / 1.402 / 255) + 128; if (V >= 255) V = 255;

	bufY[x + y * stride] = Y;
	bufU[(x / 2) + ((y / 2) * (dst_width / 2))] = U;
	bufV[(x / 2) + ((y / 2) * (dst_width / 2))] = V;
}


void cu_UVtoNV12(unsigned char *pOut, unsigned char *bufU, unsigned char *bufV, int stride, int width, int height)
{
	dim3 dimblock(8, 8);
	dim3 dimgrid(
		(width + dimblock.x - 1) / dimblock.x,
		(height + dimblock.y - 1) / dimblock.y);

	convertUVtoNV12 << <dimgrid, dimblock >> >(pOut, bufU, bufV, stride, width, height);
}

__global__ void convertUVtoNV12(unsigned char *pOut, unsigned char *bufU, unsigned char *bufV, int stride, int width, int height)
{
	unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

	pOut[(x * 2) + y * stride * 2] = bufU[x + y * width];
	pOut[((x * 2) + 1) + y * stride * 2] = bufV[x + y * width];
}
